#include "hip/hip_runtime.h"
#include "cuda_hello.h"

#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void g_cuda_hello(int* v)
{
    std::printf("Hello from Cuda! %i\n", *v);
    *v = 10;
}

__host__ int cuda_hello(int v)
{
    int h_v = v;
    int* d_v = nullptr;
    hipMalloc((void**)&d_v, sizeof(int));
    hipMemcpy(d_v, &h_v, sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

    g_cuda_hello<<<1,1>>>(d_v);

    hipMemcpy(&h_v, d_v, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
    hipFree(d_v);

    return h_v;
}